#include "hip/hip_runtime.h"
#include "Filereader.h"       // Includes Complex, vector and string
#include "hip/hip_math_constants.h"

#include <fstream>
#include <iostream>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#define FLOAT_SIZE sizeof(float)
#define BLOCK_DIM 1024

/// ---------------   DEVICE FUNCTIONS   --------------- ///
__device__ float2 dft_omega(const unsigned int k, const unsigned int n, const unsigned int N)
{
	float reciprocal = 1.0f / N;
	int exponent     = -k * n;
	float theta      = 2 * exponent * HIP_PI_F * reciprocal;

	float2 omega;
	omega.x = cosf(theta);
	omega.y = sinf(theta);

	return omega;
}

__device__ float2 idft_omega(const unsigned int k, const unsigned int n, const unsigned int N)
{
  float reciprocal = 1.0f / N;
  int exponent     = k * n;
  float theta      = 2 * exponent * HIP_PI_F * reciprocal;

  float2 omega;
  omega.x = cosf(theta);
  omega.y = sinf(theta);
  
  return omega;
}

/// @brief Compute the Discrete Fourier Transform for a single value
///
///
///  The Discrete Fourier Transform has the following general equation form:
///
///          N - 1
///   X   = [SIGMA] f  [omega_n]^-kn
///    k     n = 0   n
///
///
__global__ void dft(float* realData,  float* imagData,
                    float* realInput, float* imagInput,
                    unsigned int N)
{
  const unsigned int k = blockIdx.x * blockDim.x + threadIdx.x; // Row
  const unsigned int n = threadIdx.x;
  
  float2 omega = dft_omega(blockIdx.x, n, N);

  printf("k = %u, blockIdx.x = %u, n = %u\n", k, blockIdx.x, n);
  printf("-- omega.x = %f, omega.y = %f\n", omega.x, omega.y);
  printf("-- realInput[n] = %f, imagInput[n] = %f\n", realInput[n], imagInput[n]);

  realData[k] = (realInput[n] * omega.x) - (imagInput[n] * omega.y);
  imagData[k] = (realInput[n] * omega.y) + (imagInput[n] * omega.x);
}


/// @brief Compute the Inverse Discrete Fourier Transform for a single value
///
///
/// The Inverse Discrete Fourier Transform has the following general equation form:
///
///       1  N - 1
///  f  = -*[SIGMA] X  [omega_n]^kn
///   n   N  k = 0   k
///
///
__global__ void idft(float* realData,  float* imagData,
	                 float* realInput, float* imagInput,
	                 unsigned int N)
{
	const unsigned int n = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int k = threadIdx.x;

	float2 omega = idft_omega(k, blockIdx.x, N);

	realData[n] = (realInput[k] * omega.x) - (imagInput[k] * omega.y);
	imagData[n] = (realInput[k] * omega.y) + (imagInput[k] * omega.x);
}

/// ---------------   GLOBAL FUNCTIONS   --------------- ///

/// ----------------   HOST FUNCTIONS   ---------------- ///

/// @brief Show how the program is used when it is being used improperly
__host__ void show_program_usage()
{
  std::cout << "Improper program use detected. Please see the following for instructions:"                        << std::endl;
  std::cout << "\tgpuFFT.exe <input_file> <output_file_location>"                                                 << std::endl;
  std::cout << ""                                                                                                 << std::endl;
  std::cout << "input_file           - The input file containg the data on which to perform the FFT (must exist)" << std::endl;
  std::cout << "                       * Must have the .dat extension * "                                         << std::endl;
  std::cout << "output_file_location - The location where the output.dat file will be created (must exist)"       << std::endl;
  std::cout << ""                                                                                                 << std::endl;
  std::cout << "Please verify your inputs and try again."                                                         << std::endl;
}

__host__ unsigned int getNearestEven(unsigned int input)
{
  if (input % BLOCK_DIM == 0)
  {
    return input / BLOCK_DIM;
  }
  return input / BLOCK_DIM + 1;
}

// ====================================================
//                   EXECUTE DFT
// ====================================================

void execute_dft(std::vector<float>& realParts, std::vector<float>& imagParts)
{
  unsigned int dataSize   = (unsigned int)realParts.size();
  unsigned int deviceSize = dataSize * sizeof(float); 
  
  // Allocate the host data
  float* realResiduals;
  float* imagResiduals;

  float* hostRealData = (float*)malloc(deviceSize * dataSize);
  float* hostImagData = (float*)malloc(deviceSize * dataSize);

  for (unsigned int i = 0; i < dataSize * dataSize; ++i)
  {
	  hostRealData[i] = 0.0f;
	  hostImagData[i] = 0.0f;
  }

  float* deviceRealData;
  float* deviceImagData;
  
  // Allocate the device data
  hipMalloc(&realResiduals, deviceSize * dataSize);
  hipMalloc(&imagResiduals, deviceSize * dataSize);
  
  hipMalloc(&deviceRealData, deviceSize);
  hipMalloc(&deviceImagData, deviceSize);

  hipMemset(realResiduals, 0, deviceSize * dataSize);
  hipMemset(imagResiduals, 0, deviceSize * dataSize);

  // Copy the data from the host to the device
  hipMemcpy(deviceRealData, &realParts[0], deviceSize, hipMemcpyHostToDevice);
  hipMemcpy(deviceImagData, &imagParts[0], deviceSize, hipMemcpyHostToDevice);
  
  // Invoke the DFT
  int dimensionSize = getNearestEven(dataSize);
  dim3 gridSize(dataSize, dataSize);
  dim3 blockSize(BLOCK_DIM, BLOCK_DIM);
  dft <<< dataSize, dataSize >>> (realResiduals, imagResiduals, deviceRealData, deviceImagData, dataSize);
  
  hipDeviceSynchronize();
  
  // Copy the data from the device to the host
  hipMemcpy(hostRealData, realResiduals, deviceSize * dataSize, hipMemcpyDeviceToHost);
  hipMemcpy(hostImagData, imagResiduals, deviceSize * dataSize, hipMemcpyDeviceToHost);
  
  std::vector<float> realResult(dataSize);
  std::vector<float> imagResult(dataSize);
  
  for (unsigned int i = 0; i < dataSize; ++i)
  {
    realResult[i] = 0.0f;
    imagResult[i] = 0.0f;
    for (unsigned int j = 0; j < dataSize; ++j)
    {
		int index = (i * dataSize) + j;
		float realResidual = hostRealData[index];
		float imagResidual = hostImagData[index];

		std::cout << "(" << i << ", " << j << ") : ( " << realResidual << ", " << imagResidual << " )";
		std::cout << " -- " << index << std::endl;
		realResult[i] += realResidual;
		imagResult[i] += imagResidual;
    }
  }
  
  for (unsigned int i = 0; i < realResult.size(); ++i)
  {
    std::cout << realResult[i];
    
	  if (imagResult[i] >= 0.0f)
	  {
		  std::cout << "+";
	  }
	  std::cout << imagResult[i] << "i" << std::endl;
  }
  
  // Print the results
  // Perform cleanup
  hipFree(realResiduals);
  hipFree(imagResiduals);
  hipFree(deviceRealData);
  hipFree(deviceImagData);
 
  free(hostRealData);
  free(hostImagData);
}

// ====================================================
//                       MAIN
// ====================================================

int main(int argc, char* argv[])
{
  if (argc != 3)
  {
    show_program_usage();
    
    return EXIT_FAILURE;
  }
  
  // The first argument should be the input file.
  // Check to see if it exists
  gpuFFT::Filereader inputFileReader(argv[1]);
  if (!inputFileReader.exists())
  {
    show_program_usage();
    
    return EXIT_FAILURE;
  }
  
  std::vector<gpuFFT::Complex> inputData;
  
  std::vector<float> realParts;
  std::vector<float> imagParts;
  
  // Read the data in from the file
  inputFileReader.readFile(realParts, imagParts);
  
  for (unsigned int i = 0; i < realParts.size(); ++i)
  {
    std::cout << realParts[i];
    
	  if (imagParts[i] >= 0.0f)
	  {
		  std::cout << "+";
	  }
	  std::cout << imagParts[i] << "i" << std::endl;
  }
  
  std::cout << "Performing DFT" << std::endl;
  
  execute_dft(realParts, imagParts);
  
  return EXIT_SUCCESS;
}