#include "hip/hip_runtime.h"
#include "Filereader.h"
#include "Filewriter.h"

#include "CPUDFT.h"
#include "hip/hip_math_constants.h"

#include <fstream>
#include <iostream>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#define FLOAT_SIZE sizeof(float)
#define BLOCK_DIM 1024

/// ---------------   DEVICE FUNCTIONS   --------------- ///
__device__ float2 dft_omega(const unsigned int k, const unsigned int n, const unsigned int N)
{
	float reciprocal = 1.0f / N;
	int exponent     = -k * n;
	float theta      = 2.0f * exponent * HIP_PI_F * reciprocal;

	float2 omega;
	omega.x = cosf(theta);
	omega.y = sinf(theta);

	return omega;
}

__device__ float2 idft_omega(const unsigned int k, const unsigned int n, const unsigned int N)
{
  float reciprocal = 1.0f / N;
  int exponent     = k * n;
  float theta      = 2.0f * exponent * HIP_PI_F * reciprocal;
  
  float2 omega;
  omega.x = cosf(theta);
  omega.y = sinf(theta);
  
  return omega;
}

/// @brief Compute the Discrete Fourier Transform for a single value
///
///
///  The Discrete Fourier Transform has the following general equation form:
///
///          N - 1
///   X   = [SIGMA] f  [omega_n]^-kn
///    k     n = 0   n
///
///
__global__ void dft(float* realData,  float* imagData,
                    float* realInput, float* imagInput,
                    unsigned int N)
{
  const unsigned int k = blockIdx.x * blockDim.x + threadIdx.x; // Row
  const unsigned int n = threadIdx.x;
  
  float2 omega = dft_omega(blockIdx.x, n, N);

  realData[k] = (realInput[n] * omega.x) - (imagInput[n] * omega.y);
  imagData[k] = (realInput[n] * omega.y) + (imagInput[n] * omega.x);
}


/// @brief Compute the Inverse Discrete Fourier Transform for a single value
///
///
/// The Inverse Discrete Fourier Transform has the following general equation form:
///
///       1  N - 1
///  f  = -*[SIGMA] X  [omega_n]^kn
///   n   N  k = 0   k
///
///
__global__ void idft(float* realData,  float* imagData,
	                 float* realInput, float* imagInput,
	                 unsigned int N)
{
	const unsigned int n = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int k = threadIdx.x;

	float2 omega = idft_omega(k, blockIdx.x, N);

	realData[n] = (realInput[k] * omega.x) - (imagInput[k] * omega.y);
	imagData[n] = (realInput[k] * omega.y) + (imagInput[k] * omega.x);
}

/// ---------------   GLOBAL FUNCTIONS   --------------- ///

/// ----------------   HOST FUNCTIONS   ---------------- ///

/// @brief Show how the program is used when it is being used improperly
__host__ void show_program_usage()
{
  std::cout << "Improper program use detected. Please see the following for instructions:"                        << std::endl;
  std::cout << "\tgpuFFT.exe <input_file>"                                                                        << std::endl;
  std::cout << ""                                                                                                 << std::endl;
  std::cout << "input_file           - The input file containg the data on which to perform the FFT (must exist)" << std::endl;
  std::cout << "                       * Must have the .dat extension * "                                         << std::endl;
  std::cout << ""                                                                                                 << std::endl;
  std::cout << "Please verify your inputs and try again."                                                         << std::endl;
}

// ====================================================
//                   EXECUTE DFT
// ====================================================

void execute_gpu_dft(std::vector<float>& realParts, std::vector<float>& imagParts)
{
  unsigned int dataSize   = (unsigned int)realParts.size();
  unsigned int deviceSize = dataSize * sizeof(float); 
  
  // Allocate the host data
  float* realResiduals;
  float* imagResiduals;

  float* hostRealData = (float*)malloc(deviceSize * dataSize);
  float* hostImagData = (float*)malloc(deviceSize * dataSize);

  for (unsigned int i = 0; i < dataSize * dataSize; ++i)
  {
	  hostRealData[i] = 0.0f;
	  hostImagData[i] = 0.0f;
  }

  float* deviceRealData;
  float* deviceImagData;
  
  // Allocate the device data
  hipMalloc(&realResiduals, deviceSize * dataSize);
  hipMalloc(&imagResiduals, deviceSize * dataSize);
  
  hipMalloc(&deviceRealData, deviceSize);
  hipMalloc(&deviceImagData, deviceSize);

  hipMemset(realResiduals, 0, deviceSize * dataSize);
  hipMemset(imagResiduals, 0, deviceSize * dataSize);

  // Copy the data from the host to the device
  hipMemcpy(deviceRealData, &realParts[0], deviceSize, hipMemcpyHostToDevice);
  hipMemcpy(deviceImagData, &imagParts[0], deviceSize, hipMemcpyHostToDevice);
  
  // Invoke the DFT
  dft <<< dataSize, dataSize >>> (realResiduals, imagResiduals, deviceRealData, deviceImagData, dataSize);
  
  hipDeviceSynchronize();
  
  // Copy the data from the device to the host
  hipMemcpy(hostRealData, realResiduals, deviceSize * dataSize, hipMemcpyDeviceToHost);
  hipMemcpy(hostImagData, imagResiduals, deviceSize * dataSize, hipMemcpyDeviceToHost);
  
  std::vector<float> realResult(dataSize);
  std::vector<float> imagResult(dataSize);
  
  for (unsigned int i = 0; i < dataSize; ++i)
  {
    realResult[i] = 0.0f;
    imagResult[i] = 0.0f;
    for (unsigned int j = 0; j < dataSize; ++j)
    {
		int index = (i * dataSize) + j;
		float realResidual = hostRealData[index];
		float imagResidual = hostImagData[index];

		realResult[i] += realResidual;
		imagResult[i] += imagResidual;
    }
  }
  
  for (unsigned int i = 0; i < realResult.size(); ++i)
  {
    std::cout << realResult[i];
    
	  if (imagResult[i] >= 0.0f)
	  {
		  std::cout << "+";
	  }
	  std::cout << imagResult[i] << "i" << std::endl;
  }

  // Copy the results back into the device real and imaginary parts
  hipMemcpy(deviceRealData, &realResult[0], deviceSize, hipMemcpyHostToDevice);
  hipMemcpy(deviceImagData, &imagResult[0], deviceSize, hipMemcpyHostToDevice);

  // Invoke the IDFT
  idft <<< dataSize, dataSize >>> (realResiduals, imagResiduals, deviceRealData, deviceImagData, dataSize);

  // Copy the data from the device to the host
  hipMemcpy(hostRealData, realResiduals, deviceSize * dataSize, hipMemcpyDeviceToHost);
  hipMemcpy(hostImagData, imagResiduals, deviceSize * dataSize, hipMemcpyDeviceToHost);

  realResult.clear();
  imagResult.clear();

  realResult.resize(dataSize, 0.0f);
  imagResult.resize(dataSize, 0.0f);

  for (unsigned int i = 0; i < dataSize; ++i)
  {
	  realResult[i] = 0.0f;
	  imagResult[i] = 0.0f;
	  for (unsigned int j = 0; j < dataSize; ++j)
	  {
		  int index = (i * dataSize) + j;
		  float realResidual = hostRealData[index];
		  float imagResidual = hostImagData[index];

		  realResult[i] += realResidual / dataSize;
		  imagResult[i] += imagResidual / dataSize;
	  }
  }

  for (unsigned int i = 0; i < realResult.size(); ++i)
  {
	  std::cout << realResult[i];

	  if (imagResult[i] >= 0.0f)
	  {
		  std::cout << "+";
	  }
	  std::cout << imagResult[i] << "i" << std::endl;
  }
  
  // Free the device-memory
  hipFree(realResiduals);
  hipFree(imagResiduals);
  hipFree(deviceRealData);
  hipFree(deviceImagData);
 
  // Free the host-memory
  free(hostRealData);
  free(hostImagData);
}

/// @brief  Executes the CPU-based Discrete Fourier Transform
///         (and inverse Discrete Fourier Transform)
///
/// @param[in] real    The real parts of the input
/// @param[in] imag    The imaginary parts of the input

// TODO: ADD OUTPUTS FOR TIMINGS (DFT and IDFT)
void execute_cpu_dft(std::vector<float>& real, std::vector<float>& imag)
{
	// Create the CPU-based DFT object
	gpuFFT::CPUDFT cpuDFT(real, imag);

	// Create vectors to contain the transformed data
	std::vector<float> transformedReal;
	std::vector<float> transformedImag;

	// Perform the Discrete Fourier Transform
	// (note that the DFT operation does not affect the original input)
	cpuDFT.dft(transformedReal, transformedImag);

	// Write out the DFT results to a file
	gpuFFT::Filewriter cpuDFTWriter("cpu_dft_output.dat");
	cpuDFTWriter.write(transformedReal, transformedImag);

	// Create the CPU-based DFT object for the inverse DFT
	gpuFFT::CPUDFT cpuIDFT(transformedReal, transformedImag);

	// Perform the Inverse Discrete Fourier Transform
	// (note that the IDFT operation does not affect the original input)
	cpuIDFT.idft(transformedReal, transformedImag);

	// Write out the IDFT results to a file
	gpuFFT::Filewriter writer("cpu_output.dat");
	writer.write(transformedReal, transformedImag);
}

// ====================================================
//                       MAIN
// ====================================================

int main(int argc, char* argv[])
{
  if (argc != 2)
  {
    show_program_usage();
    
    return EXIT_FAILURE;
  }
  
  // The first argument should be the input file.
  // Check to see if it exists
  gpuFFT::Filereader inputFileReader(argv[1]);
  if (!inputFileReader.exists())
  {
    show_program_usage();
    
    return EXIT_FAILURE;
  }
  
  // TODO: CLEAN UP AND DOCUMENT
  std::vector<float> realParts;
  std::vector<float> imagParts;
  
  // Read the data in from the file
  inputFileReader.readFile(realParts, imagParts);
  
  // Perform some output [DEBUG - TO BE REMOVED]
  for (unsigned int i = 0; i < realParts.size(); ++i)
  {
    std::cout << realParts[i];
    
	  if (imagParts[i] >= 0.0f)
	  {
		  std::cout << "+";
	  }
	  std::cout << imagParts[i] << "i" << std::endl;
  }
  
  // ===========================================
  // Execute the GPU DFT
  // ===========================================
  execute_gpu_dft(realParts, imagParts);

  // ===========================================
  // Execute the CPU DFT
  // ===========================================
  execute_cpu_dft(realParts, imagParts);  

  return EXIT_SUCCESS;
}